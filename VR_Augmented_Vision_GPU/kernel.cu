#include "hip/hip_runtime.h"
#include "Kernal.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"
#include "math.h"

#include <stdio.h>
#include <stdlib.h>

unsigned char number_of_cameras;
unsigned int frame_width, frame_height;

unsigned int projected_frame_width, projected_frame_height;
unsigned char* projected_frame;//dont need a second buffer because memcpy is thread agnostic, so worst case a single pixel is corrupted

enum camera_names{
	top_frame = 0,
	bottom_frame = 1,
	front_frame = 2,
	left_frame = 3,
	right_frame = 4,
	back_frame = 5,
};

typedef struct{
	unsigned char* frame_0;//these two serve as frame buffers
	unsigned char* frame_1;
	unsigned char selected_frame;//this is used by the thread to get pixel data from the current frame. Must be in range 0 to number_of buffer frames-1
} Frame_Info;

Frame_Info* frame_array;//pointer stored on host memory, which points to device memory

#define THREADS_PER_BLOCK_MAX 1024 //note this this usually can't be used since the number of available registers is usually the lower bound
#define THREADS_PER_BLOCK_USED 512

#define BLOCKS_MAX 65535 
unsigned int BLOCKS_USED;

//allocate space for the array of images, and the buffer images for each camera, and the final projected image
//also set some global constants
int allocate_frames(unsigned char arg_number_of_cameras, 
					 unsigned int arg_frame_width, unsigned int arg_frame_height, 
					 unsigned int arg_projected_frame_width, unsigned int arg_projected_frame_height)
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	//save these variables
	number_of_cameras = arg_number_of_cameras;
	frame_width = arg_frame_width;
	frame_height = arg_frame_height;
	projected_frame_width = arg_projected_frame_width;
	projected_frame_height = arg_projected_frame_height;
	
	cudaStatus = hipMalloc(&frame_array, arg_number_of_cameras*sizeof(Frame_Info));//allocate space for frame array
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return 1;
	}

	for (unsigned char i = 0; i < arg_number_of_cameras; ++i){
		//setup frame info locally (point to device allocations)
		Frame_Info camera_frame_info;
		hipMalloc(&camera_frame_info.frame_0, arg_frame_width*arg_frame_height * 3 * sizeof(unsigned char));
		hipMalloc(&camera_frame_info.frame_1, arg_frame_width*arg_frame_height * 3 * sizeof(unsigned char));
		camera_frame_info.selected_frame = 0;
		//copy over local frame info to device memory
		cudaStatus = hipMemcpy(&frame_array[i], &camera_frame_info, sizeof(Frame_Info), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc failed!");
			return 1;
		}
	}

	cudaStatus = hipMalloc(&projected_frame, arg_projected_frame_width*arg_projected_frame_height * 4 * sizeof(unsigned char));//allocate the projected frame
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return 1;
	}

	//compute the number of blocks to assign 1 pixel per thread
	BLOCKS_USED = (projected_frame_width*projected_frame_height) / THREADS_PER_BLOCK_USED;
	if (BLOCKS_USED > BLOCKS_MAX){
		printf("Require %d blocks, greater than BLOCKS_MAX.\n", BLOCKS_USED);//TODO in this case, each thread should do multiple pixels
		return 1;
	}

	return 0;
}

//given a pointer to an image on the host memory, copy to the currently unused frame buffer for that frame in the device memory, and update the selected frame indicator
//don't need syncronization like in non-gpu code, because we're not changing the pointer, but writing inplace to the buffer, which is thread safe. Worst case 1 corrupted pixel.
//updating selected frame must be atomic though
void copy_new_frame(unsigned char camera, unsigned char* image_data){
	//since we can't dereference device memory on host code
	Frame_Info frame;
	hipMemcpy(&frame, &frame_array[camera], sizeof(Frame_Info), hipMemcpyDeviceToHost);//get addresses in device memory for the images

	switch (frame.selected_frame){
		case 0:
			hipMemcpy(frame.frame_1, image_data, frame_width*frame_height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
			frame.selected_frame = 1;
			break;
		case 1:
			hipMemcpy(frame.frame_0, image_data, frame_width*frame_height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
			frame.selected_frame = 0;
			break;
	}

	hipMemcpy(&(frame_array[camera].selected_frame), &frame.selected_frame, sizeof(unsigned char), hipMemcpyHostToDevice);//TODO atomic
}

//copy the generated projected frame stored on the gpu to the cpu memory
void read_projected_frame(unsigned char*  host_projection_frame){
	hipError_t cudaStatus = hipMemcpy(host_projection_frame, projected_frame, projected_frame_width*projected_frame_height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
}

//this is the function that is parralelized
//each handles a single pixel on the projection screen
__global__ void Project_to_Screen(unsigned int projected_frame_height, unsigned int projected_frame_width, 
								  unsigned int frame_width, unsigned int frame_height,
								  Frame_Info* frame_array, unsigned char* projected_frame)
{
	unsigned int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int j = thread_num / projected_frame_width;//pixel row(height)
	unsigned int i = thread_num % projected_frame_width;//pixel column(width)

	//http://stackoverflow.com/questions/34250742/converting-a-cubemap-into-equirectangular-panorama
	//inverse mapping

	double u, v; //Normalised texture coordinates, from 0 to 1, starting at lower left corner
	double phi, theta; //Polar coordinates

	//convert x,y cartesian to u,v polar

	//Rows start from the bottom
	v = 1 - ((double)j / projected_frame_height);
	theta = v * HIP_PI;

	//Columns start from the left
	u = ((double)i / projected_frame_width);
	phi = u * 2 * HIP_PI;


	//convert polar to 3d vector
	double x, y, z; //Unit vector
	x = sin(phi) * sin(theta) * -1;
	y = cos(theta);
	z = cos(phi) * sin(theta) * -1;

	double xa, ya, za;
	double a;

	a = fmax(fmax(abs(x), abs(y)), abs(z));

	//Vector Parallel to the unit vector that lies on one of the cube faces
	xa = x / a;
	ya = y / a;
	za = z / a;

	unsigned char pixel[3];
	int xPixel, yPixel;

	//while (1)
	{
		if (xa == 1)
		{
			//Right
			xPixel = (int)((((za + 1.0) / 2.0) - 1.0) * frame_width);
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			switch (frame_array[right_frame].selected_frame){
				case 0:
					pixel[0] = frame_array[right_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[right_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[right_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
				case 1:
					pixel[0] = frame_array[right_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[right_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[right_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
			}
			/*pixel[0] = 0;
			pixel[1] = 0;
			pixel[2] = 255;//red*/
		}
		else if (xa == -1)
		{
			//Left
			xPixel = (int)((((za + 1.0) / 2.0)) * frame_width);
			if (xPixel >= frame_width){
				xPixel = frame_width - 1;
			}
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			switch (frame_array[left_frame].selected_frame){
				case 0:
					pixel[0] = frame_array[left_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[left_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[left_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
				case 1:
					pixel[0] = frame_array[left_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[left_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[left_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
			}
			/*pixel[0] = 0;
			pixel[1] = 255;
			pixel[2] = 255;//yellow*/
		}
		else if (ya == -1)
		{
			//Up
			xPixel = (int)((((xa + 1.0) / 2.0)) * frame_width);
			yPixel = (int)((((za + 1.0) / 2.0) - 1.0) * frame_height);
			//flip vertical
			yPixel = (frame_height - 1) - abs(yPixel);

			switch (frame_array[top_frame].selected_frame){
				case 0:
					pixel[0] = frame_array[top_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[top_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[top_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
				case 1:
					pixel[0] = frame_array[top_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[top_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[top_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
			}
			/*pixel[0] = 0;
			pixel[1] = 60;
			pixel[2] = 255;//orange*/
		}
		else if (ya == 1)
		{
			//Down
			xPixel = (int)((((xa + 1.0) / 2.0)) * frame_width);
			yPixel = (int)((((za + 1.0) / 2.0)) * frame_height);
			//flip vertical
			yPixel = (frame_height - 1) - abs(yPixel);

			switch (frame_array[bottom_frame].selected_frame){
				case 0:
					pixel[0] = frame_array[bottom_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[bottom_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[bottom_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
				case 1:
					pixel[0] = frame_array[bottom_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[bottom_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[bottom_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
			}
			/*pixel[0] = 255;
			pixel[1] = 0;
			pixel[2] = 0;//blue*/
		}
		else if (za == 1)
		{
			//Front
			xPixel = (int)((((xa + 1.0) / 2.0)) * frame_width);
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			switch (frame_array[front_frame].selected_frame){
				case 0:
					pixel[0] = frame_array[front_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[front_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[front_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
				case 1:
					pixel[0] = frame_array[front_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[front_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[front_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
			}
			/*pixel[0] = 150;
			pixel[1] = 150;
			pixel[2] = 150;//grey*/
		}
		else if (za == -1)
		{
			//Back
			xPixel = (int)((((xa + 1.0) / 2.0) - 1.0) * frame_width);
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			switch (frame_array[back_frame].selected_frame){
				case 0:
					pixel[0] = frame_array[back_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[back_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[back_frame].frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
				case 1:
					pixel[0] = frame_array[back_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 0];
					pixel[1] = frame_array[back_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 1];
					pixel[2] = frame_array[back_frame].frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * 3) + 2];
					break;
			}
			/*pixel[0] = 150;
			pixel[1] = 0;
			pixel[2] = 0;//light blue*/
		}
		else
		{
			printf("Unknown face, something went wrong");
		}

		//converting to RGBA from BGR, with max A
		projected_frame[((j*projected_frame_width + i) * 4) + 0] = pixel[2];
		projected_frame[((j*projected_frame_width + i) * 4) + 1] = pixel[1];
		projected_frame[((j*projected_frame_width + i) * 4) + 2] = pixel[0];
		projected_frame[((j*projected_frame_width + i) * 4) + 3] = 0xFF;
	}
}

void cuda_run(){
	Project_to_Screen << <BLOCKS_USED, THREADS_PER_BLOCK_USED >> >(projected_frame_height, projected_frame_width, frame_width, frame_height, frame_array, projected_frame);

	//fuck error checking and blocking, WE'RE GOING FAST
	// Check for any errors launching the kernel
	/*hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching kernal!\n", cudaStatus);
	}*/
}