#include "hip/hip_runtime.h"
#include "Kernal.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"
#include "math.h"

#include <stdio.h>
#include <stdlib.h>

/*-----Structs and Enums----*/

//enum for each ViewPoint
enum viewpoint_names{
	top_view = 0,
	bottom_view = 1,
	front_view = 2,
	left_view = 3,
	right_view = 4,
	back_view = 5,
};

#define NUMBER_OF_VIEWPOINTS 6
unsigned int frame_width, frame_height;
unsigned int projected_frame_width, projected_frame_height;

typedef struct{
	unsigned char* left;
	unsigned char* right;
} Projected_Frame;

#define projected_frame_bytes 4 //the projected frames must be 4bytes per pixel because Oculus 
Projected_Frame projected_frame_host;//the host's copy of the projected frame. pinned memory
Projected_Frame projected_frame;//dont need a second buffer because memcpy is thread agnostic, so worst case a single pixel is corrupted

#define cube_frame_bytes 3
//image and buffer for each cube face
//TODO is is actaully better to have this buffer and switch?
typedef struct{
	unsigned char* frame_0;//frames used for frame buffer
	unsigned char* frame_1;
	unsigned char selected_frame;//frame current in use from buffer
} Frame_Pointer;

//eye views for each cube face
typedef struct{
	Frame_Pointer left;
	Frame_Pointer right;
} Frame;

//array of mats of the cube faces
Frame* cube_faces;//TODO test speed of this being on host and pointing to device, vs all on device

#define THREADS_PER_BLOCK_MAX 1024 //note this this usually can't be used since the number of available registers is usually the lower bound
#define THREADS_PER_BLOCK_USED 512

#define BLOCKS_MAX 65535 
unsigned int BLOCKS_USED;

//allocate space for the array of images, and the buffer images for each camera, and the final projected image
//also set some global constants
//return the pointer to the host's copy of projected frame, or NULL for any error
void* allocate_frames(unsigned int arg_frame_width, unsigned int arg_frame_height, 
					 unsigned int arg_projected_frame_width, unsigned int arg_projected_frame_height)
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		return NULL;
	}

	//save these variables
	frame_width = arg_frame_width;
	frame_height = arg_frame_height;
	projected_frame_width = arg_projected_frame_width;
	projected_frame_height = arg_projected_frame_height;

	cudaStatus = hipMalloc(&cube_faces, sizeof(Frame)*NUMBER_OF_VIEWPOINTS);
	if (cudaStatus != hipSuccess) {
		printf("Failed to malloc cube_faces!\n");
		return NULL;
	}

	for (unsigned char i = 0; i < NUMBER_OF_VIEWPOINTS; ++i){
		//setup frame info locally (point to device allocations)
		Frame cube_face;
		hipMalloc(&cube_face.left.frame_0, arg_frame_width*arg_frame_height * cube_frame_bytes * sizeof(unsigned char));
		hipMalloc(&cube_face.left.frame_1, arg_frame_width*arg_frame_height * cube_frame_bytes * sizeof(unsigned char));
		cube_face.left.selected_frame = 0;
		hipMalloc(&cube_face.right.frame_0, arg_frame_width*arg_frame_height * cube_frame_bytes * sizeof(unsigned char));
		hipMalloc(&cube_face.right.frame_1, arg_frame_width*arg_frame_height * cube_frame_bytes * sizeof(unsigned char));
		cube_face.right.selected_frame = 0;
		//copy over local frame info to device memory
		cudaStatus = hipMemcpy(&cube_faces[i], &cube_face, sizeof(cube_face), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("Failed to copy Frame_Info!\n");
			return NULL;
		}
	}

	//allocate the host's projected frame as pinned memory
	cudaStatus = hipHostMalloc(&projected_frame_host.left, arg_projected_frame_width*arg_projected_frame_height * projected_frame_bytes * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		printf("Failed to malloc projected_frame_host left!\n");
		return NULL;
	}
	cudaStatus = hipHostMalloc(&projected_frame_host.right, arg_projected_frame_width*arg_projected_frame_height * projected_frame_bytes * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		printf("Failed to malloc projected_frame_host right!\n");
		return NULL;
	}
	//allocate the gpu's projected frame
	cudaStatus = hipMalloc(&projected_frame.left, arg_projected_frame_width*arg_projected_frame_height * projected_frame_bytes * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		printf("Failed to malloc projected_frame left!\n");
		return NULL;
	}
	cudaStatus = hipMalloc(&projected_frame.right, arg_projected_frame_width*arg_projected_frame_height * projected_frame_bytes * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		printf("Failed to malloc projected_frame right!\n");
		return NULL;
	}

	//compute the number of blocks to assign 1 pixel per thread
	BLOCKS_USED = (projected_frame_width*projected_frame_height) / THREADS_PER_BLOCK_USED;
	if (BLOCKS_USED > BLOCKS_MAX){
		printf("Require %d blocks, greater than BLOCKS_MAX.\n", BLOCKS_USED);//TODO in this case, each thread should do multiple pixels
		return NULL;
	}

	return &projected_frame_host;
}

//given a pointer to an image on the host memory, copy to the currently unused frame buffer for that frame in the device memory, and update the selected frame indicator
//don't need syncronization like in non-gpu code, because we're not changing the pointer, but writing inplace to the buffer, which is thread safe. Worst case 1 corrupted pixel.
//updating selected frame must be atomic though
void copy_new_frame(unsigned char view, bool left_eye, unsigned char* image_data, unsigned int image_x, unsigned int image_y, unsigned int slice_width, unsigned int slice_height){
	//since we can't dereference device memory on host code
	//extra the struct and image pointers from array
	Frame_Pointer frame;
	if (left_eye){
		hipMemcpy(&frame, &cube_faces[view].left, sizeof(Frame_Pointer), hipMemcpyDeviceToHost);//get addresses in device memory for the images
	}else{
		hipMemcpy(&frame, &cube_faces[view].right, sizeof(Frame_Pointer), hipMemcpyDeviceToHost);
	}

	//copy in the slice to the cube frame
	//and update the selected frame
	switch (frame.selected_frame){
		case 0:
			//copy the given image (a slice) into the frame
			for (unsigned int slice_y = 0; slice_y < slice_height; ++slice_y){
				hipMemcpyAsync(frame.frame_1 + (frame_width*sizeof(unsigned char)*cube_frame_bytes)*image_y + image_x,
							    image_data    + (slice_width*sizeof(unsigned char)*cube_frame_bytes)*slice_y,
								slice_width*sizeof(unsigned char)*cube_frame_bytes, hipMemcpyHostToDevice);
				image_y++;
			}
			frame.selected_frame = 1;
			break;
		case 1:
			for (unsigned int slice_y = 0; slice_y < slice_height; ++slice_y){
				hipMemcpyAsync(frame.frame_0 + (frame_width*sizeof(unsigned char)*cube_frame_bytes)*image_y + image_x,
								image_data    + (slice_width*sizeof(unsigned char)*cube_frame_bytes)*slice_y,
								slice_width*sizeof(unsigned char)*cube_frame_bytes, hipMemcpyHostToDevice);
				image_y++;
			}
			frame.selected_frame = 0;
			break;
	}

	//copy back the selected frame update
	if (left_eye){
		hipMemcpy(&(cube_faces[view].left.selected_frame), &frame.selected_frame, sizeof(unsigned char), hipMemcpyHostToDevice);//TODO atomic
	}
	else{
		hipMemcpy(&(cube_faces[view].right.selected_frame), &frame.selected_frame, sizeof(unsigned char), hipMemcpyHostToDevice);//TODO atomic
	}
}

//copy the generated projected frame stored on the gpu to the cpu memory
//use pinned host memory, non-waited async copy (because like kernal, longer to check if copied than to actually copy), and multiple streams (in form of memcpy2D)
//TODO can this be optimized so that only the necissary eye is copied?
void read_projected_frame(){
	hipMemcpy2DAsync(projected_frame_host.left,  projected_frame_width*sizeof(unsigned char) * projected_frame_bytes, projected_frame.left,  projected_frame_width*sizeof(unsigned char) * projected_frame_bytes, projected_frame_width*sizeof(unsigned char) * projected_frame_bytes, projected_frame_height, hipMemcpyDeviceToHost);
	hipMemcpy2DAsync(projected_frame_host.right, projected_frame_width*sizeof(unsigned char) * projected_frame_bytes, projected_frame.right, projected_frame_width*sizeof(unsigned char) * projected_frame_bytes, projected_frame_width*sizeof(unsigned char) * projected_frame_bytes, projected_frame_height, hipMemcpyDeviceToHost);
}

//helper function for getting pixel data from frame
__device__ __forceinline__ void Get_Pixel_From_Frame(Frame_Pointer frame, int yPixel, int xPixel, int frame_width,
													 unsigned char* pixel_out)
{
	switch (frame.selected_frame){
		case 0:
			pixel_out[0] = frame.frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * cube_frame_bytes) + 0];
			pixel_out[1] = frame.frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * cube_frame_bytes) + 1];
			pixel_out[2] = frame.frame_0[((abs(yPixel)*frame_width + abs(xPixel)) * cube_frame_bytes) + 2];
			break;
		case 1:
			pixel_out[0] = frame.frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * cube_frame_bytes) + 0];
			pixel_out[1] = frame.frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * cube_frame_bytes) + 1];
			pixel_out[2] = frame.frame_1[((abs(yPixel)*frame_width + abs(xPixel)) * cube_frame_bytes) + 2];
			break;
	}
}

//this is the function that is parralelized
//each handles a single pixel on the projection screen
__global__ void Project_to_Screen(unsigned int projected_frame_height, unsigned int projected_frame_width, 
								  unsigned int frame_width, unsigned int frame_height,
								  Frame* frame_array, Projected_Frame projected_frame)
{
	unsigned int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int j = thread_num / projected_frame_width;//pixel row(height)
	unsigned int i = thread_num % projected_frame_width;//pixel column(width)

	//http://stackoverflow.com/questions/34250742/converting-a-cubemap-into-equirectangular-panorama
	//inverse mapping

	double u, v; //Normalised texture coordinates, from 0 to 1, starting at lower left corner
	double phi, theta; //Polar coordinates

	//convert x,y cartesian to u,v polar

	//Rows start from the bottom
	v = 1 - ((double)j / projected_frame_height);
	theta = v * HIP_PI;

	//Columns start from the left
	u = ((double)i / projected_frame_width);
	phi = u * 2 * HIP_PI;


	//convert polar to 3d vector
	double x, y, z; //Unit vector
	x = sin(phi) * sin(theta) * -1;
	y = cos(theta);
	z = cos(phi) * sin(theta) * -1;

	double xa, ya, za;
	double a;

	a = fmax(fmax(abs(x), abs(y)), abs(z));

	//Vector Parallel to the unit vector that lies on one of the cube faces
	xa = x / a;
	ya = y / a;
	za = z / a;

	//handle both eyes and output images
	unsigned char pixel_left[cube_frame_bytes];
	unsigned char pixel_right[cube_frame_bytes];
	int xPixel, yPixel;

	//while (1)
	{
		if (xa == 1)
		{
			//Right
			xPixel = (int)((((za + 1.0) / 2.0) - 1.0) * frame_width);
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);
			Get_Pixel_From_Frame(frame_array[right_view].left, yPixel, xPixel, frame_width, pixel_left);
			Get_Pixel_From_Frame(frame_array[right_view].right, yPixel, xPixel, frame_width, pixel_right);
		}
		else if (xa == -1)
		{
			//Left
			xPixel = (int)((((za + 1.0) / 2.0)) * frame_width);
			if (xPixel >= frame_width){
				xPixel = frame_width - 1;
			}
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			Get_Pixel_From_Frame(frame_array[left_view].left, yPixel, xPixel, frame_width, pixel_left);
			Get_Pixel_From_Frame(frame_array[left_view].right, yPixel, xPixel, frame_width, pixel_right);
		}
		else if (ya == -1)
		{
			//Up
			xPixel = (int)((((xa + 1.0) / 2.0)) * frame_width);
			yPixel = (int)((((za + 1.0) / 2.0) - 1.0) * frame_height);
			//flip vertical
			yPixel = (frame_height - 1) - abs(yPixel);

			Get_Pixel_From_Frame(frame_array[top_view].left, yPixel, xPixel, frame_width, pixel_left);
			Get_Pixel_From_Frame(frame_array[top_view].right, yPixel, xPixel, frame_width, pixel_right);
		}
		else if (ya == 1)
		{
			//Down
			xPixel = (int)((((xa + 1.0) / 2.0)) * frame_width);
			yPixel = (int)((((za + 1.0) / 2.0)) * frame_height);
			//flip vertical
			yPixel = (frame_height - 1) - abs(yPixel);

			Get_Pixel_From_Frame(frame_array[bottom_view].left, yPixel, xPixel, frame_width, pixel_left);
			Get_Pixel_From_Frame(frame_array[bottom_view].right, yPixel, xPixel, frame_width, pixel_right);
		}
		else if (za == 1)
		{
			//Front
			xPixel = (int)((((xa + 1.0) / 2.0)) * frame_width);
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			Get_Pixel_From_Frame(frame_array[front_view].left, yPixel, xPixel, frame_width, pixel_left);
			Get_Pixel_From_Frame(frame_array[front_view].right, yPixel, xPixel, frame_width, pixel_right);
		}
		else if (za == -1)
		{
			//Back
			xPixel = (int)((((xa + 1.0) / 2.0) - 1.0) * frame_width);
			yPixel = (int)((((ya + 1.0) / 2.0)) * frame_height);

			Get_Pixel_From_Frame(frame_array[back_view].left, yPixel, xPixel, frame_width, pixel_left);
			Get_Pixel_From_Frame(frame_array[back_view].right, yPixel, xPixel, frame_width, pixel_right);
		}
		else
		{
			printf("Unknown face, something went wrong");
		}

		//converting to RGBA from BGR, with max A
		projected_frame.left[((j*projected_frame_width + i) * projected_frame_bytes) + 0] = pixel_left[2];
		projected_frame.left[((j*projected_frame_width + i) * projected_frame_bytes) + 1] = pixel_left[1];
		projected_frame.left[((j*projected_frame_width + i) * projected_frame_bytes) + 2] = pixel_left[0];
		projected_frame.left[((j*projected_frame_width + i) * projected_frame_bytes) + 3] = 0xFF;

		projected_frame.right[((j*projected_frame_width + i) * projected_frame_bytes) + 0] = pixel_right[2];
		projected_frame.right[((j*projected_frame_width + i) * projected_frame_bytes) + 1] = pixel_right[1];
		projected_frame.right[((j*projected_frame_width + i) * projected_frame_bytes) + 2] = pixel_right[0];
		projected_frame.right[((j*projected_frame_width + i) * projected_frame_bytes) + 3] = 0xFF;
	}
}

void cuda_run(){
	Project_to_Screen << <BLOCKS_USED, THREADS_PER_BLOCK_USED >> >(projected_frame_height, projected_frame_width, frame_width, frame_height, cube_faces, projected_frame);

	//fuck error checking and blocking, WE'RE GOING FAST
	//we actually don't need it because it takes FAR longer to check if kernal is finished then to actually run kernal

	// Check for any errors launching the kernel
	/*hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching kernal!\n", cudaStatus);
	}*/
}